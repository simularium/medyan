
#include <hip/hip_runtime.h>

//------------------------------------------------------------------
//  **MEDYAN** - Simulation Package for the Mechanochemical
//               Dynamics of Active Networks, v3.1
//
//  Copyright (2015-2016)  Papoian Lab, University of Maryland
//
//                 ALL RIGHTS RESERVED
//
//  See the MEDYAN web page for more information:
//  http://www.medyan.org
//------------------------------------------------------------------
#ifdef HYBRID_NLSTENCILLIST
#include "HybridNeighborListImpl.h"

#include "Bead.h"
#include "Filament.h"
#include "Cylinder.h"
#include "Bubble.h"
#include "BoundaryElement.h"

#include "GController.h"
#include "MathFunctions.h"
#include "CUDAcommon.h"

using namespace mathfunc;

short HybridCylinderCylinderNL::totalhybridNL = 0;
void HybridCylinderCylinderNL::updateallcylinderstobin() {
    for(auto cyl:Cylinder::getCylinders())
        updatebin(cyl);
}

void HybridCylinderCylinderNL::assignallcylinderstobin() {
    for(auto cyl:Cylinder::getCylinders())
        assignbin(cyl);
/*    std::cout<<"H Total number of bins "<< _binGrid->getBins().size()<<endl;
    for(auto bin:_binGrid->getBins()){
        std::cout<<bin->getCylinders().size()<<" ";
    }
    std::cout<<endl;*/
}

void HybridCylinderCylinderNL::assignbin(Cylinder* cyl){
    Bin* _bin;
    try {_bin = getBin(cyl->coordinate);}
    catch (exception& e) {
        cout << e.what() << endl;
        exit(EXIT_FAILURE);
    }
    _bin->addCylinder(cyl);
    cyl->_hbinvec.push_back(_bin);
}

void HybridCylinderCylinderNL::unassignbin(Cylinder* cyl, Bin* bin){
    bin->removeCylinder(cyl);
}

void HybridCylinderCylinderNL::updatebin(Cylinder *cyl){
    Bin* _bin;
//    std::cout<<coordinate[0]<<" "<<coordinate[1]<<" "<<coordinate[2]<<endl;
    try {_bin = getBin(cyl->coordinate);}
    catch (exception& e) {
        cout << e.what();
        cyl->printSelf();
        exit(EXIT_FAILURE);
    }
    if(_bin != cyl->_hbinvec.at(_ID)) {
#ifdef CHEMISTRY
        auto oldBin = cyl->_hbinvec.at(_ID);
        auto newBin = _bin;
#endif
        //remove from old compartment, add to new
        oldBin->removeCylinder(cyl);
        cyl->_hbinvec.at(_ID) = newBin;
        _bin->addCylinder(cyl);
    }
}

void HybridCylinderCylinderNL::generateConnections() {
    for(size_t i=0U; i<_grid[0]; ++i) {

        for(size_t j=0U; j<_grid[1]; ++j) {

            for(size_t k=0U; k<_grid[2]; ++k) {
                vector<size_t> indices{i,j,k};
                Bin *target = getBin(indices);//defined in this file.

                vector<double> coordinates =
                        {indices[0] * _binSize[0] + _binSize[0] / 2,
                         indices[1] * _binSize[1] + _binSize[1] / 2,
                         indices[2] * _binSize[2] + _binSize[2] / 2};
                target->setCoordinates(coordinates);
                int stencilcount = 0;

                //Go through all neighbors to get the neighbors list
                for(int ii: {-1,0,1}){
                    for(int jj: {-1,0,1}){
                        for(int kk: {-1,0,1}){
                            //Consider the target bin itself as a neighbor.
                            stencilcount++;
                            int iprime = i+ii;
                            int jprime = j+jj;
                            int kprime = k+kk;
                            if(iprime<0 or iprime==int(_grid[0]) or jprime<0 or
                               jprime==int(_grid[1]) or kprime<0 or
                               kprime==int(_grid[2]))
                                continue;
                            vector<size_t> currentIndices{size_t(iprime), size_t
                                    (jprime), size_t(kprime)};
                            Bin *neighbor = getBin(currentIndices);
                            target->addNeighbour(neighbor);
                            target->stencilID.push_back(stencilcount-1);
                        }
                    }
                }
            }
        }
    }


    /*for(size_t i=0U; i<_grid[0]; ++i) {

        for (size_t j = 0U; j < _grid[1]; ++j) {

            for (size_t k = 0U; k < _grid[2]; ++k) {
                vector<size_t> indices{i, j, k};
                Bin *target = getBin(indices);
                std::cout << "Target " << target->coordinates()[0] << " " <<
                          target->coordinates()[1] << " " <<
                          target->coordinates()[2] << " " << endl;
                std::cout<<"Bin size "<<_binSize[0]<<endl;
                for (int ii: {-1, 0, 1}) {
                    for (int jj: {-1, 0, 1}) {
                        for (int kk: {-1, 0, 1}) {
                            int iprime = i + ii;
                            int jprime = j + jj;
                            int kprime = k + kk;
                            if (iprime < 0 or iprime == int(_grid[0]) or jprime < 0 or
                                jprime == int(_grid[1]) or kprime < 0 or
                                kprime == int(_grid[2]))
                                continue;
                            vector<size_t> currentIndices{size_t(iprime), size_t
                                    (jprime), size_t(kprime)};
                            Bin *neighbor = getBin(currentIndices);
                            std::cout << "Neighbor " << neighbor->coordinates()[0]
                                      << " " <<
                                      neighbor->coordinates()[1] << " " <<
                                      neighbor->coordinates()[2] << " " << endl;
                        }
                    }
                }
            }
        }
    }*/
}

void HybridCylinderCylinderNL::initializeBinGrid() {

//    //Initial parameters of system
    auto _nDim = SysParams::Geometry().nDim;
    double searchdist = 1.125 * (sqrt(_largestrMaxsq));
//    std::cout<<"H searchdist "<<searchdist<<" rMax "<<sqrt(_largestrMaxsq)<<endl;
    _binSize = {searchdist, searchdist, searchdist};
    if(_nDim >=1) {
        _size.push_back(int(SysParams::Geometry().NX * SysParams::Geometry()
                .compartmentSizeX));
        if( (_size[0]) % int(_binSize[0]) ==0)
            _grid.push_back(_size[0]/_binSize[0]);
        else
            _grid.push_back(_size[0]/_binSize[0] + 1);
    }
    if (_nDim >= 2) {
        _size.push_back(int(SysParams::Geometry().NY * SysParams::Geometry()
                .compartmentSizeY));
        if( (_size[1]) % int(_binSize[1]) ==0)
            _grid.push_back(_size[1]/_binSize[1]);
        else
            _grid.push_back(_size[1]/_binSize[1] + 1);
    }
    if (_nDim == 3) {
        _size.push_back(int(SysParams::Geometry().NZ * SysParams::Geometry()
                .compartmentSizeZ));
        if( (_size[2]) % int(_binSize[2]) ==0)
            _grid.push_back(_size[2]/_binSize[2]);
        else
            _grid.push_back(_size[2]/_binSize[2] + 1);
    }

    //Check that grid and compartmentSize match nDim
    if((_nDim == 3 &&
        _grid[0] != 0 && _grid[1] != 0 && _grid[2]!=0 &&
        _binSize[0] != 0 &&
        _binSize[1] != 0 &&
        _binSize[2] != 0)){
    }
    else {
        cout << "Bin parameters for CylinderCylinderNeighborLists are invalid. Exiting." <<
             endl;
        exit(EXIT_FAILURE);
    }
    int size = 1;
    for(auto x: _grid) {
        if(x != 0) size*=x;
    }
    //Set the instance of this grid with given parameters
    _binGrid = new BinGrid(size, _ID, _binSize);
    //Create connections based on dimensionality
    generateConnections();
}

//You need a vector of all grids so you can loop through and update respective coordinates.
Bin* HybridCylinderCylinderNL::getBin(const vector<double> &coords) {
    //Check if out of bounds
    size_t index = 0;
    size_t i = 0;
    for(auto x: coords)
    {
        //Flatten the coordinates to 1D, get integer index
        if(i == 0) {
            if(x < 0 || x >= (_binSize[0] * _grid[0])) {
                cout<<"get Bin coords x"<<endl;
                throw OutOfBoundsException();
            }

            index += int(x / _binSize[0]);
        }
        else if(i == 1) {
            if(x < 0 || x >= (_binSize[1] * _grid[1])) {
                cout<<"get Bin coords y"<<endl;
                throw OutOfBoundsException();
            }

            index += int(x / _binSize[1]) * _grid[0];
        }
        else {
            if(x < 0 || x >= (_binSize[2] * _grid[2])) {
                cout<<"get Bin coords z"<<endl;
                throw OutOfBoundsException();
            }

            index += int(x / _binSize[2]) * _grid[0] * _grid[1];
        }
        i++;
    }

    try {
        return _binGrid->getBin(index);
    }
    catch (exception& e){
        cout << "Bad bin access at..." << endl;
        cout << "Bin index = " << index << endl;
        cout << "Coords = " << coords[0] << " " << coords[1] << " " << coords[2] << endl;
        throw NaNCoordinateException();
    }
}

Bin* HybridCylinderCylinderNL::getBin(const vector<size_t> &indices) {
    size_t index = 0;
    size_t i = 0;
    for(auto x: indices)
    {
        //Flatten the indices to 1D
        if(i == 0) {
            if(x >= _grid[0]) {
                cout<<"get Bin x"<<endl;
                throw OutOfBoundsException();
            }

            index += x;
        }
        else if(i == 1) {
            if(x >= _grid[1]) {
                cout<<"get Bin y"<<endl;
                throw OutOfBoundsException();
            }

            index += x * _grid[0];
        }
        else {
            if(x >= _grid[2]) {
                cout << "get Bin z" << endl;
                throw OutOfBoundsException();
            }

            index += x * _grid[0] * _grid[1];
        }

        i++;
    }
    try {
        return _binGrid->getBin(index);
    }
    catch (exception& e){
        cout << "Bad Bin access at..." << endl;
        cout << "Bin index = " << index << endl;
        cout << "Indices = " << indices[0] << " " << indices[1] << " " << indices[2] << endl;
        throw NaNCoordinateException();
    }
}

void HybridCylinderCylinderNL::updateNeighborsbin(Cylinder* currcylinder, bool runtime){
    //clear existing neighbors of currcylinder from all neighborlists
    for(int idx = 0; idx < totaluniquefIDpairs; idx++) {
        int countbounds = _rMaxsqvec[idx].size();
        for (int idx2 = 0; idx2 < countbounds; idx2++) {
            auto HNLID = HNLIDvec[idx][idx2];
            _list4mbinvec[HNLID][currcylinder].clear();
        }
    }
    //get necessary variables
    auto binvec = currcylinder->_hbinvec;//The different hybrid bins that this cylinder
    // belongs to.
    //Check if the cylinder has been assigned a bin. If not, assign.
    if(binvec.size()<=_ID)
        assignbin(currcylinder);
    binvec = currcylinder->_hbinvec;
    //get parent bin corresponding to this hybrid neighbor list.
    auto parentbin =  binvec.at(_ID);
    //get neighboring bins
    vector<Bin*> _neighboringBins = binvec.at(_ID)//Get the bin that belongs to the
                    // current binGrid of interest for this NL.
                                                    ->getNeighbours();
    double *coord = CUDAcommon::getSERLvars().coord;
    auto cylindervec = CUDAcommon::getSERLvars().cylindervec;
    auto cylinderpointervec = CUDAcommon::getSERLvars().cylinderpointervec;
    int cindex = currcylinder->_dcIndex;
    cylinder c = cylindervec[cindex];

    //
    int ncyls2 = 0;
    int tcyl2 = 0;
    int nbincount = 0;
    auto nbinstencil = parentbin->stencilID;// A standard templated numbering of
    // neighboring bins is implemented i.e. based on position w.r.t. bin of interest,
    // neighboring bins are given a particular ID.nbinstencil stores the set of such
    // neighbors that is close to bin of interest. Bins close to the boundary will have
    // < 27 elements in the stencilID vector.
    short ftype1 = c.type; //cylinder type and filament type is one and the
    // same.
    float _largestrMax = sqrt(_largestrMaxsq);
    for (auto &bin : _neighboringBins) {
            bool isbinneeded = _binGrid->iswithincutoff(c.coord,
                                                        parentbin->coordinates(),
                                                        nbinstencil.at(nbincount),
                                                        _largestrMax);
            nbincount++;
            if (isbinneeded) {
                auto cindicesvec = bin->getcindices();
                int numneighbors = cindicesvec.size();
                for (int iter = 0; iter < numneighbors; iter++) {
                    int ncindex = cindicesvec[iter];
                    cylinder ncylinder = cylindervec[ncindex];
                    short ftype2 = ncylinder.type;
//                    //Don't add the same cylinder
//                    if (c.ID == ncylinder.ID) continue;
                    // Testing if a half neighborlist will be stable
                    if(c.ID <= ncylinder.ID) continue;
                    //Don't add if belonging to same parent
                    if (c.filamentID == ncylinder.filamentID) {
                        auto distsep = fabs(c.filamentposition - ncylinder.filamentposition);
                        //if not cross filament, check if not neighboring
                        if (distsep <= 2) continue;
                    }

                    //Loop through all the distance bounds and add to neighborlist
                    for (int idx = 0; idx < totaluniquefIDpairs; idx++) {
                        int countbounds = _rMaxsqvec[idx].size();
                        auto fpairs = _filamentIDvec[idx].data();
                        //Check for cylinder filament types
                        if (ftype1 < ftype2) {
                            if (ftype1 != fpairs[0] || ftype2 != fpairs[1])continue;
                        }
                        else if (ftype1 != fpairs[1] || ftype2 != fpairs[0]) continue;
                        double dist = twoPointDistancesquared(c.coord, ncylinder.coord);
                        if (dist < _smallestrMinsq || dist > _largestrMaxsq) continue;
                        for (int idx2 = 0; idx2 < countbounds; idx2++) {
                            //Dont add if ID is more than cylinder for half-list
//                            if (!_fullstatusvec[idx][idx2] && c.ID <= ncylinder.ID) continue;
                            //Dont add if not within range
                            if (dist > _rMaxsqvec[idx][idx2] ||
                                dist < _rMinsqvec[idx][idx2])
                                continue;
                            short HNLID = HNLIDvec[idx][idx2];
                            //If we got through all of this, add it!
                            Cylinder *Ncylinder = cylinderpointervec[ncindex];
                            _list4mbinvec[HNLID][currcylinder].push_back(Ncylinder);
                            //if runtime, add to other list as well if full
/*                            if (runtime && _fullstatusvec[idx][idx2]) {
                                _list4mbinvec[HNLID][Ncylinder].push_back(currcylinder);
                            }*/
                        }
                    }
                }
            }
    }
}

vector<Cylinder*> HybridCylinderCylinderNL::getNeighborsstencil(short HNLID, Cylinder*
                                                                cylinder) {

    return _list4mbinvec[HNLID][cylinder];
}

void HybridCylinderCylinderNL::addNeighbor(Neighbor* n) {

    //return if not a cylinder!
    Cylinder* cylinder;
    if(!(cylinder = dynamic_cast<Cylinder*>(n))) return;

    //update neighbors
    updateNeighborsbin(cylinder, true);
}

void HybridCylinderCylinderNL::removeNeighbor(Neighbor* n) {

    Cylinder* cylinder;
    if(!(cylinder = dynamic_cast<Cylinder*>(n))) return;
    for(int idx = 0; idx < totaluniquefIDpairs; idx++) {
        int countbounds = _rMaxsqvec[idx].size();
        for (int idx2 = 0; idx2 < countbounds; idx2++) {
            auto HNLID = HNLIDvec[idx][idx2];
/*            std::cout << "Removing neighbors of cylinder with cindex " <<
                          cylinder->_dcIndex<<" and ID "<<cylinder->getID() << " from NL " << HNLID << endl;*/
            //Remove from NeighborList
            _list4mbinvec[HNLID].erase(cylinder);
            //Remove from bin
            Bin *bin = cylinder->_hbinvec.at(_ID);
            unassignbin(cylinder, bin);
            //remove from other lists
//            std::cout << "Removed from cylinders ";
            for (auto it = _list4mbinvec[HNLID].begin();
                 it != _list4mbinvec[HNLID].end(); it++) {
                auto cit = find(it->second.begin(), it->second.end(), cylinder);
                {
                    if (cit != it->second.end()) {
                        it->second.erase(cit);
//                        std::cout << it->first->getID() << " ";
                    }
                }
            }
//            std::cout<<endl;
        }
    }

}

void HybridCylinderCylinderNL::reset() {

    //loop through all neighbor keys
    for(int idx = 0; idx < totalhybridNL; idx++) {
        _list4mbinvec[idx].clear();
//        std::cout<<"Hybrid rmin rmax "<<_rMinsqvec[idx]<<" "<<_rMaxsqvec[idx]<<endl;
    }

    /*chrono::high_resolution_clock::time_point mins, mine;
    mins = chrono::high_resolution_clock::now();*/
    //check and reassign cylinders to different bins if needed.
    updateallcylinderstobin();
    _binGrid->updatecindices();
    for(auto cylinder: Cylinder::getCylinders()) {
        updateNeighborsbin(cylinder);
//        for (int idx = 0; idx < totalhybridNL; idx++) {
//            tot[idx] += _list4mbinvec[idx][cylinder].size();
//        }
    }
//    std::cout<<endl;
//    for(int idx = 0; idx < totalhybridNL; idx++)
//        std::cout<<"reset HybridNLSTENCILLIST size "<<" "<<tot[idx]<<endl;
/*    mine= chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_sten(mine - mins);
    std::cout<<"Hybrid NLSTEN reset time "<<elapsed_sten.count()<<endl;*/

    //Check if HNLID = 1 is symmetric
/*    short HNLID = 1;
    int idx = 0; int idx2 = _rMaxsqvec[idx].size()-1;
    auto _list4mbin = _list4mbinvec[HNLID];
    std::cout<<"map size = " << _list4mbin.size()<<endl;
    std::cout << "max_size = " << _list4mbin.max_size() <<endl;
    for(auto cylinder: Cylinder::getCylinders()) {
        auto neighbors = _list4mbin[cylinder];
        auto cylinderbin = cylinder->_hbinvec[0];
        auto cylbincoord = cylinderbin->coordinates();
        for(auto ncylinder:neighbors){
            auto ncylinderbin = ncylinder->_hbinvec[0];
            auto ncylbincoord = ncylinderbin->coordinates();
            //look for cylinder in the neighbor list of ncylinder
            auto ncylinderneighbors = _list4mbin[ncylinder];
            std::cout<<"neighborvec size "<<ncylinderneighbors.size()<<" capacity "
                     <<ncylinderneighbors.capacity()<<" max_size "<<ncylinderneighbors
                    .max_size()<<endl;
            if(find(ncylinderneighbors.begin(),ncylinderneighbors.end(),cylinder) ==
                    ncylinderneighbors.end()){
                std::cout<<" cylinder "<<cylinder->getID()<<" from bin "
                        ""<<cylinderbin<<" "
                        "coordinates "<<cylbincoord[0]<<" "<<cylbincoord[1]<<" "
                        ""<<cylbincoord[2]<<" has neighbor cylinder "<<ncylinder->getID()
                         <<" from bin "<<ncylinderbin<<" coordinates "
                        ""<<ncylbincoord[0]<<" "<<ncylbincoord[1]<<" "
                        ""<<ncylbincoord[2]<<endl;
                std::cout<<"But neighbor cylinder does not have cylinder in it's "
                        "neighbors list. Check algorithm."<<endl;
            }
        }
    }*/
}
#endif